
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <chrono>
using namespace std;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void gpuKernel(int *a) {
    int location =  blockIdx.x * blockDim.x + threadIdx.x;
    for (size_t i = 0; i < 100; ++i) 
        a[location] += location/i;
}

int main(int argc, char** argv) {
    // begin time
    std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();
    const size_t size = 32*1024*1024; // 8 MP img
    const uint n_tries = 50;

    int *a_host;
    a_host = (int*) malloc(size*sizeof(int)*n_tries);
    for (int i = 0; i < size*n_tries; ++i) 
        a_host[i] = -i;


    int *a_dev;

    // gpuErrchk(cudaHostAlloc((void** ) &a, size*sizeof(int), cudaHostAllocDefault));
    for (int i = 0; i < n_tries; ++i ) {
        std::cout << "Try " << i << std::endl;
        gpuErrchk(hipMalloc((void** )&a_dev, size*sizeof(int)));
        gpuErrchk(hipMemcpy(a_dev, a_host+size*i, size*sizeof(int), hipMemcpyHostToDevice));
        gpuKernel<<<size/32, 32>>>(a_dev);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
        gpuErrchk(hipMemcpy(a_host+size*i, a_dev, size*sizeof(int), hipMemcpyDeviceToHost));
        gpuErrchk(hipFree(a_dev));
    }

    std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
    std::cout << "Duration = " << std::chrono::duration_cast<std::chrono::milliseconds>(end - begin).count() <<std::endl;
}