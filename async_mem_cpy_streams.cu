
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <chrono>
using namespace std;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void fastTask(int *a) {
    int location =  blockIdx.x * blockDim.x + threadIdx.x;
    for (size_t i = 0; i < 100; ++i) 
        a[location] += location/i;
}

int main(int argc, char** argv) {
    // begin time
    std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();
    const size_t size = 32*1024*1024;
    const uint n_tries = 50;
    int *a_host;
    int **a_dev;
    
    a_host = (int *) malloc(size*sizeof(int)*n_tries);
    a_dev = (int **) malloc(3*sizeof(int*));
    for (int i = 0; i < 3; ++i) {
        gpuErrchk(hipMalloc((void** )&a_dev[i], size*sizeof(int)));
    }

    for (int i = 0; i < size*n_tries; ++i) 
        a_host[i] = -i;

    hipStream_t s[3];
    gpuErrchk(hipStreamCreate(&s[0]));
    gpuErrchk(hipStreamCreate(&s[1]));
    gpuErrchk(hipStreamCreate(&s[2]));
    // gpuErrchk(cudaHostAlloc((void** ) &a, size*sizeof(int), cudaHostAllocDefault));
    for (int i = 0; i < n_tries+2; ++i ) {
        std::cout << "Try " << i << std::endl;
        // gpuErrchk(cudaMemcpy(a_dev, a_host+size*i, size*sizeof(int), cudaMemcpyHostToDevice));
        // fastTask<<<size/32, 32>>>(a_dev);
        // gpuErrchk(cudaPeekAtLastError());
        // gpuErrchk(cudaDeviceSynchronize());
        // gpuErrchk(cudaMemcpy(a_host+size*i, a_dev, size*sizeof(int), cudaMemcpyDeviceToHost));
        if (i >=0 && i < n_tries) {
            hipStream_t& HD_stream = s[i%3];
            gpuErrchk(hipMemcpyAsync(a_dev[i%3], a_host+size*i, size*sizeof(int), hipMemcpyHostToDevice, HD_stream));
        }
        if (i>1 && i < n_tries+1) {
            hipStream_t& process_stream = s[(i-1)%3];
            fastTask<<<size/32, 32, 0, process_stream>>>(a_dev[(i-1)%3]);
        }
        if (i>2 && i < n_tries + 2) {
            hipStream_t& DH_stream = s[(i-2)%3];
            gpuErrchk(hipMemcpyAsync(a_host+size*(i-2), a_dev[(i-2)%3], size*sizeof(int), hipMemcpyDeviceToHost, DH_stream));
        }

    }
    for (int i = 0; i < 3; ++i) {
        gpuErrchk(hipFree(a_dev[i]));
    }
    free(a_host);
    free(a_dev);

    std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
    std::cout << "Duration = " << std::chrono::duration_cast<std::chrono::milliseconds>(end - begin).count() <<std::endl;
}